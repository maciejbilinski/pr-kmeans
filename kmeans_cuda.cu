#include "hip/hip_runtime.h"
#include <iostream>
#include <map>
#include <cmath>

#include "constants.h"
#include "point.h"
#include "common_func.h"

int THREADS_PER_BLOCK;

#define getIndex(row, column) row * DATA_DIMENSIONS + column

int getGridSize(int n){
    return (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}

// inicjalizuje centroidy na podstawie losowo wybranych identyfikatorów znajdujących się w number
__global__ void initCentroids(int* i, int* numbers, float* centroids, float* data){
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < DATA_DIMENSIONS) {
        centroids[getIndex(*i, j)] = data[getIndex(numbers[*i], j)];
    }
}

// metoda `distance(Point*, Point*)` przepisana na działanie na GPU
__device__ void gpuDistance(float* p1, float* p2, float* output) {
    float sum = 0.0;
    for (int i = 0; i < DATA_DIMENSIONS; i++) {
        float diff = *(p1 + i) - *(p2 + i);
        sum += (diff * diff);
    }
    *output = std::sqrt(sum);
}

// metoda `gpuGetCentroid(Point, Point*)` przepisana na działanie na GPU
__device__ void gpuGetCentroid(float* p, float* centroids, int* output) {
    int closestCentroid = 0;
    float minDist;
    gpuDistance(p, centroids, &minDist);
    for (int j = 1; j < LABELS_NUMBER; j++) {
        float dist;
        gpuDistance(p, centroids + j * DATA_DIMENSIONS, &dist);
        if (dist < minDist) {
            minDist = dist;
            closestCentroid = j;
        }
    }
    *output = closestCentroid;
}

// wstępnie oblicza wartości centroidów
__global__ void calculateCentroids(float* centroids, float* data, float* newCentroids, int* clusterSizes){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < DATA_LINES) {
        int cluster;
        gpuGetCentroid(data + i*DATA_DIMENSIONS, centroids, &cluster);
        for (int j = 0; j < DATA_DIMENSIONS; j++) {
            atomicAdd(&newCentroids[getIndex(cluster, j)], data[getIndex(i, j)]);
        }
        atomicAdd(&clusterSizes[cluster], 1);
    }
}

// dokańcza obliczanie wartości centroidów
__global__ void finishCalculateCentroids(float* newCentroids, int* clusterSizes){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < LABELS_NUMBER) {
        if (clusterSizes[i] > 0) {
            for (int j = 0; j < DATA_DIMENSIONS; j++) {
                newCentroids[getIndex(i, j)] /= clusterSizes[i];
            }
        }
    }
}

// przepisuje wartości z `newCentroids` do `centroids`
__global__ void rewriteCentroids(int* i, float* newCentroids, float* centroids){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < DATA_DIMENSIONS) {
        centroids[getIndex(*i, j)] = newCentroids[getIndex(*i, j)];
    }
}

__global__ void checkCentroidsChanged(float* centroids, float* newCentroids, bool* centroidsChanged) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < LABELS_NUMBER) {
        float dist;
        gpuDistance(centroids + i, newCentroids + i, &dist);
        if(dist > 0.0){
            *centroidsChanged = true;
        }
    }
}

void kMeansCUDA(Point* cpuCentroids, Point* cpuData, int MAX_ITERATIONS) {
    // kopiowanie danych na GPU
    float* centroids;
    float* data;
    hipMalloc((void**)&centroids, LABELS_NUMBER * DATA_DIMENSIONS * sizeof(float));
    hipMalloc((void**)&data, DATA_LINES * DATA_DIMENSIONS * sizeof(Point));
    for (int i = 0; i < DATA_LINES; i++) {
        hipMemcpy(&data[i*DATA_DIMENSIONS], cpuData[i].values, DATA_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);
    }

    // losowanie początkowych wartości
    int* cpuNumbers = getRandomDataIndexes();
    int* numbers;
    hipMalloc((void**)&numbers, LABELS_NUMBER * sizeof(int));
    hipMemcpy(numbers, cpuNumbers, LABELS_NUMBER * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < LABELS_NUMBER; i++) {
        int* x;
        hipMalloc((void**)&x, sizeof(int));
        hipMemcpy(x, &i, sizeof(int), hipMemcpyHostToDevice);
        initCentroids<<<getGridSize(DATA_DIMENSIONS), THREADS_PER_BLOCK>>>(x, numbers, centroids, data);
        hipDeviceSynchronize();
        hipFree(x);
    }
    hipFree(numbers);
    delete[] cpuNumbers;

    // inicjalizacja pamięci
    int* clusterSizes;
    float* newCentroids;
    hipMalloc((void**)&clusterSizes, LABELS_NUMBER * sizeof(int));
    hipMalloc((void**)&newCentroids, LABELS_NUMBER * DATA_DIMENSIONS * sizeof(float));

    int* clusterSizes0 = (int*)calloc(LABELS_NUMBER, sizeof(int));
    float* newCentroids0 = (float*)calloc(LABELS_NUMBER * DATA_DIMENSIONS, sizeof(float));

    // główna pętla algorytmu
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        if (VERBOSE)
            std::cout << "Iteration " << iter << std::endl;

        // czyszczenie pamięci - ustalanie wartości na zero
        hipMemcpy(clusterSizes, clusterSizes0, LABELS_NUMBER * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(newCentroids, newCentroids0, LABELS_NUMBER * DATA_DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);

        // poszukiwanie nowych centroidów
        calculateCentroids<<<getGridSize(DATA_LINES), THREADS_PER_BLOCK>>>(centroids, data, newCentroids, clusterSizes);
        hipDeviceSynchronize();
        finishCalculateCentroids<<<getGridSize(LABELS_NUMBER), THREADS_PER_BLOCK>>>(newCentroids, clusterSizes);
        hipDeviceSynchronize();

        // opcjonalny mechanizm wczesnego zatrzymywania algorytmu
        if (EARLY_STOP) {
            bool centroidsChanged = false;
            bool* gpuCentroidsChanged;

            hipMalloc(&gpuCentroidsChanged, sizeof(bool));
            hipMemcpy(gpuCentroidsChanged, &centroidsChanged, sizeof(bool), hipMemcpyHostToDevice);

            checkCentroidsChanged<<<getGridSize(LABELS_NUMBER), THREADS_PER_BLOCK>>>(centroids, newCentroids, gpuCentroidsChanged);
            hipDeviceSynchronize();
            hipMemcpy(&centroidsChanged, gpuCentroidsChanged, sizeof(bool), hipMemcpyDeviceToHost);
            
            hipFree(gpuCentroidsChanged);

            if (!centroidsChanged) {
                break;
            }
        }

        // przepisywanie nowych centroidów
        for (int i = 0; i < LABELS_NUMBER; i++) {
            int* x;
            hipMalloc((void**)&x, sizeof(int));
            hipMemcpy(x, &i, sizeof(int), hipMemcpyHostToDevice);
            rewriteCentroids<<<getGridSize(DATA_DIMENSIONS), THREADS_PER_BLOCK>>>(x, newCentroids, centroids);
            hipDeviceSynchronize();
            hipFree(x);
        }
    }

    // zwalnianie niepotrzebnej pamięci
    hipFree(newCentroids);
    hipFree(clusterSizes);
    free(newCentroids0);
    free(clusterSizes0);

    // kopiowanie z powrotem do CPU
    for (int i = 0; i < LABELS_NUMBER; i++) {
        hipMemcpy(cpuCentroids[i].values, centroids + i * DATA_DIMENSIONS, DATA_DIMENSIONS * sizeof(float), hipMemcpyDeviceToHost);
    }

    // przypisanie centoidom etykiety na podstawie największej ilości blisko znajdujących się punktów
    // ten etap tak naprawdę nie należy do głównej części algorytmu, ale wykonujemy go dla ewaluacji algorytmu
    if(SET_CENTROID_LABELS){
        std::map<std::string, int>* labelsCount = new std::map<std::string, int>[LABELS_NUMBER];
        for (int i = 0; i < DATA_LINES; i++) {
            int cluster = getCentroid(cpuData[i], cpuCentroids);
            if (labelsCount[cluster].count(cpuData[i].label) == 1) {
                labelsCount[cluster][cpuData[i].label]++;
            } else {
                labelsCount[cluster].insert({cpuData[i].label, 1});
            }
        }

        for (int i = 0; i < LABELS_NUMBER; i++) {
            int maxCount = 0;
            for (auto it = labelsCount[i].begin(); it != labelsCount[i].end(); ++it) {
                if (it->second > maxCount) {
                    maxCount = it->second;
                    cpuCentroids[i].label = it->first;
                }
            }
        }
        delete[] labelsCount;
    }

    // zwalnianie pamięci na GPU
    hipFree(centroids);
    hipFree(data);
}

int main(int argc, char* argv[]) {
    if(argc >= 3){
        THREADS_PER_BLOCK = atoi(argv[2]);
    }else{
        THREADS_PER_BLOCK = 256;
    }
    
    std::cout << runProgram(kMeansCUDA, atoi(argv[1])) << std::endl;
    return 0;
}